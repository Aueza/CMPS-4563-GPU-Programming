// Ethan Saenz
// CMPS 4563 - GPU Programming
// Assignment 1 - CUDA Programming
// This program computes the product of two arrays using CUDA.

const int N = 4096; // Size of the arrays.


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

// Compute array product: C = A * B
// Occurs in device (GPU) global memory (kernel).
__global__
void arrProductKernel(const int *A_d, const int *B_d, int *C_d, int N){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < N){
        C_d[i] = A_d[i] * B_d[i];
    }
}

int main(){
    // allocating host (CPU) arrays.
    int A[N], B[N], C[N];

    // initializing host arrays.
    for(int i = 0; i < N; i++){
        A[i] = i;
        B[i] = i + 4095;
    }

    // checking array initialization.
    printf("Array A: \n");
    printf("A[0] = %d, A[4095] = %d\n", A[0], A[4095]);
    printf("\nArray B: \n");
    printf("B[0] = %d, B[4095] = %d\n", B[0], B[4095]);

    // allocating device (GPU) arrays.
    int *A_d, *B_d, *C_d;
    const int size = N * sizeof(int);

    // allocating memory (global) on the device (GPU).
    // (address of ptr to be allocated, size of allocated object).
    hipMalloc((void**) &A_d, size);
    hipMalloc((void**) &B_d, size);
    hipMalloc((void**) &C_d, size);

    // copying host (CPU) arrays to device (GPU) arrays.
    // (destination, source, size, direction).
    hipMemcpy(A_d, A, size, hipMemcpyHostToDevice);
    hipMemcpy(B_d, B, size, hipMemcpyHostToDevice);

    // defining the computation grid.
    dim3 dimGrid(32, 1); // 32 blocks.
    dim3 dimBlock(256, 1); // 256 threads per block.

    // launching the kernel with the grid.
    arrProductKernel<<<dimGrid, dimBlock>>>(A_d, B_d, C_d, N);

    // copying device (GPU) array to host (CPU) array.
    hipMemcpy(C, C_d, size, hipMemcpyDeviceToHost);

    // printing the results.
    printf("\nArray C: \n");
    printf("C[0] = %d, C[4095] = %d", C[0], C[4095]);

    // printing the summation of the results.
    int sum = 0;
    for(int i = 0; i < N; i++){
        sum += C[i];
    }
    printf("\n\nSum of all elements in C = %d\n", sum);

    // freeing device (GPU) memory.
    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);
    return 0;
}
