//**************************************************************
// Assignment #3
// Name: Ethan Saenz and Isaiah Corrales 
// GPU Programming Date: (M/D/Y)
//**************************************************************
// This program 
//**************************************************************


//**************************************************************
// FOR ALL FUNCTIONS function Name::MethodName()
// Parameters: List them here and comment
// A discussion of what the method/function does and required
// parameters as well as return value.
//**************************************************************


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

const int N = 8192;
const int blockSize = 1024;
const int gridSize = 4;
const float pi = M_PI;

__global__
void FFT(float *Ad, float *Bd, float *Cd, float *Dd, int N, const float pi){
	// Getting global thread ID.
	int threadId = blockIdx.x * blockDim.x + threadIdx.x;

	// Each thread works with 2 indices.
	int index1 = threadId;
	int index2 = threadId + (N / 2);

	// Computing the FFT for the first index. (0 - 4095)
	float twiddleReal = cos((2 * pi * (threadId)) / N);
	float twiddleImag = -(sin((2 * pi * (threadId)) / N));
	Cd[index1] = (Ad[index1] * twiddleReal) + (-1 * (Bd[index1] * twiddleImag)); // real
    Dd[index1] = (Ad[index1] * twiddleImag) + (Bd[index1] * twiddleReal); // imaginary


    // Computing the FFT for the second index. (4096 - 8191)
    //twiddleReal = cos((2 * pi * (threadId)) / N);
    //twiddleImag = -(sin((2 * pi * (threadId)) / N));
    Cd[index2] = (Ad[index2] * twiddleReal) + (-1 * (Bd[index2] * twiddleImag)); // real
    Dd[index2] = (Ad[index2] * twiddleImag) + (Bd[index2] * twiddleReal); // imaginary
	
}

int main(){
	// Allocate arrays in main memory.
	float A[N] = {3.6, 2.9, 5.6, 4.8, 3.3, 5.9, 5, 4.3};
	float B[N] = {2.6, 6.3, 4, 9.1, 0.4, 4.8, 2.6, 4.1};
	float C[N], D[N];

	// Define pointers for global memory.
	float *Ad, *Bd, *Cd, *Dd;
	int size = N * sizeof(float);

	// Allocate array space in global memory.
	hipMalloc((void**) &Ad, size);
	hipMalloc((void**) &Bd, size);
	hipMalloc((void**) &Cd, size);
	hipMalloc((void**) &Dd, size);

	// Copy main memory arrays to global memory.
	hipMemcpy(Ad, A, size, hipMemcpyHostToDevice);
	hipMemcpy(Bd, B, size, hipMemcpyHostToDevice);

	// Defining the computational grid.
	dim3 dimGrid(gridSize);
	dim3 dimBlock(blockSize);

	// Invoking the kernel with the defined computation grid.
	FFT<<<dimGrid, dimBlock>>>(Ad, Bd, Cd, Dd, N, pi);

	// Copying result arrays from global to main memory.
	hipMemcpy(C, Cd, size, hipMemcpyDeviceToHost);
	hipMemcpy(D, Dd, size, hipMemcpyDeviceToHost);

	// Printing the first seven results.
	printf("\n\nTOTAL PROCESSED SAMPLES: %d\n", N);
	printf("===============================================\n");
	for(int i = 0; i < 8; i++){
		printf("XR[%d]: %f 		XI[%d]: %f\n", i, C[i], i, D[i]);
		printf("===============================================\n");
	}


}